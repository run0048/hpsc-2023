
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void init(int *bucket, int range) {
  int i = threadIdx.x;
  if(i>=range) return;
  bucket[i] = 0;
}

__global__ void add(int *bucket, int *key,  int n) {
  int i = threadIdx.x;
  if(i>=n) return;
  bucket[key[i]]++;
}

int main() {
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key,n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  int *bucket;
  hipMallocManaged(&bucket,range*sizeof(int));
  init<<<1,range>>>(bucket,range);
  hipDeviceSynchronize();
  
  add<<<1,n>>>(bucket,key,n);
  hipDeviceSynchronize();
  /*
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }*/

 
  for (int i=0, j=0; i<range; i++) {
    for (; bucket[i]>0; bucket[i]--) {
      key[j++] = i;
    }
  

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(bucket);
  }
}
